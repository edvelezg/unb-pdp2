
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

const int N = 16;
const int blocksize = 16;

__global__
void add_matrix_gpu( float* a, float *b, float *c, int N )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int index = i + j*N;
    if ( i < N && j < N )
        c[index] = a[index] + b[index];
}



int main() {

    float *a = new float[N*N];
    float *b = new float[N*N];
    float *c = new float[N*N];

    for ( int i = 0; i < N*N; ++i ) {
        a[i] = 1.0f; b[i] = 3.5f;
    }

    float *ad, *bd, *cd;
    const int size = N*N*sizeof(float);

    hipMalloc( (void**)&ad, size );
    hipMalloc( (void**)&bd, size );
    hipMalloc( (void**)&cd, size );
    hipMemcpy( ad, a, size, hipMemcpyHostToDevice );
    hipMemcpy( bd, b, size, hipMemcpyHostToDevice );


    dim3 dimBlock( blocksize, blocksize );
    dim3 dimGrid(N/blocksize,N/blocksize);

    add_matrix_gpu<<<dimGrid, dimBlock>>>( ad, bd, cd, N );

    hipMemcpy( c, cd, size, hipMemcpyDeviceToHost );

    hipFree( ad ); hipFree( bd ); hipFree( cd );

    for ( int i = 0; i < N; ++i ) {
        for ( int j = 0; j < N; ++j ) {
            int index = i + j*N;
//          if ( index%1000==0 )
            cout << c[index] << " ";
        }
        cout << endl; 
    }

    delete[] a; 
    delete[] b;
    delete[] c;
    return EXIT_SUCCESS;
}

