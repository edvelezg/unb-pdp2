#include "hip/hip_runtime.h"
/*
 * This is a basic example of how to use the CUDPP library.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include "cutil.h"
#include "cudpp.h"

__global__
void mult_matrix_by_vector( float* a, float *b, float *c, int N )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float Cvalue = 0;
    int index;

    /*
    * Each thread will perform the dot product between the row of the matrix 
    * and the vector that is being multiplied. 
    */
    if ( i < N )
    {
        for ( int e = 0; e < N; ++e )
        {
            index = e + i*N;
            Cvalue += a[index]*b[e];
        }
        c[i] = Cvalue;
    }
}

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    runTest( argc, argv);
    CUT_EXIT(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
    CUT_DEVICE_INIT(argc, argv);

    unsigned int numElements = 32; // number of elements 
    unsigned int memSize = sizeof( float) * numElements; // size of the memory

    // allocate host memory
    float* h_frequencies = (float*) malloc( memSize); // allocating input data
    char* h_symbols = (char*) malloc( memSize); // allocating input data

    // initalizing the memory with the elements
    for (unsigned int i = 0; i < numElements; ++i) 
    {
		h_frequencies[i] = (float) (i+1);
		printf("i = %f\n", h_frequencies[i]);
    }
	
	// allocating symbolic data
    for (unsigned int i = 0; i < numElements; ++i) 
    {
		h_symbols[i] = 'A' + (char)i; // (rand() & 0xf);
		printf("i = %c\n", h_symbols[i]);
    }


    // allocate device memory for frequencies
    float* d_frequencies; // frequencies
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_frequencies, memSize));
    // copy host memory to device
    CUDA_SAFE_CALL( hipMemcpy( d_frequencies, h_frequencies, memSize,
                                hipMemcpyHostToDevice) );

    // allocate device memory for exclusive scan output
    float* d_exclusiveScan; // exclusive scan output
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_exclusiveScan, memSize));

    // Initialize the CUDPP Library
    CUDPPHandle theCudpp;
    cudppCreate(&theCudpp);

    CUDPPConfiguration config;
    config.op = CUDPP_ADD;
    config.datatype = CUDPP_FLOAT;
    config.algorithm = CUDPP_SCAN;
    config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE;
    
    CUDPPHandle scanplan = 0;
    CUDPPResult result = cudppPlan(theCudpp, &scanplan, config, numElements, 1, 0);  

    if (CUDPP_SUCCESS != result)
    {
        printf("Error creating CUDPPPlan\n");
        exit(-1);
    }

    // Run the scan
    cudppScan(scanplan, d_exclusiveScan, d_frequencies, numElements);

    // allocate mem for the result on host side
    float* h_odata = (float*) malloc( memSize);
    // copy result from device to host
    CUDA_SAFE_CALL( hipMemcpy( h_odata, d_exclusiveScan, memSize,
                                hipMemcpyDeviceToHost) );

	for(size_t i = 0; i < numElements; ++i)
	{
		printf("res: %f\n", h_odata[i]);
	}
	
    result = cudppDestroyPlan(scanplan);
    if (CUDPP_SUCCESS != result)
    {
        printf("Error destroying CUDPPPlan\n");
        exit(-1);
    }

    // shut down the CUDPP library
    cudppDestroy(theCudpp);
    
    free( h_frequencies);
    free( h_odata);
    // free( reference);
    CUDA_SAFE_CALL(hipFree(d_frequencies));
    CUDA_SAFE_CALL(hipFree(d_exclusiveScan));
}
